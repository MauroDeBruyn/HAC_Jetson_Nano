#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


#define WIDTH 128
#define HEIGHT 128
#define NUM_STREAMS 4

__global__ 
void imageProcessingKernel(int *image, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int index = y * width + x;
        // Simulated image processing task: setting pixel value to 1
        image[index] = 1;
    }
}

int main()
{
    int *d_image[NUM_STREAMS];
    int *h_image[NUM_STREAMS];
    hipStream_t stream[NUM_STREAMS];
    
    size_t size = WIDTH * HEIGHT * sizeof(int);
    
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipMalloc((void **)&d_image[i], size);
        hipHostMalloc(&h_image[i], size, hipHostMallocDefault);
        
        for (int j = 0; j < WIDTH * HEIGHT; ++j) {
            h_image[i][j] = 0; // Initialize image data
        }
        
        hipStreamCreate(&stream[i]);
    }
    
    dim3 blockSize(16, 16);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (HEIGHT + blockSize.y - 1) / blockSize.y);
    
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipMemcpyAsync(d_image[i], h_image[i], size, hipMemcpyHostToDevice, stream[i]);
    }
    
    for (int i = 0; i < NUM_STREAMS; ++i) {
        imageProcessingKernel<<<gridSize, blockSize, 0, stream[i]>>>(d_image[i], WIDTH, HEIGHT);
    }
    
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipMemcpyAsync(h_image[i], d_image[i], size, hipMemcpyDeviceToHost, stream[i]);
    }
    
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamSynchronize(stream[i]);
    }
    
    // Print result (just printing non-zero elements for simplicity)
    for (int i = 0; i < NUM_STREAMS; ++i) {
        printf("Stream %d:\n", i);
        for (int j = 0; j < WIDTH * HEIGHT; ++j) {
            if (h_image[i][j] != 0) {
                printf("(%d, %d): %d\n", j % WIDTH, j / WIDTH, h_image[i][j]);
            }
        }
    }
    
    // Free memory and destroy streams
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipFree(d_image[i]);
        hipHostFree(h_image[i]);
        hipStreamDestroy(stream[i]);
    }
    
    printf("\nDone\n");
    
    return 0;
}
